#include <hip/hip_runtime.h>

#include <cmath>

// Define the size of the output bitmap
__device__ bool isPointInTriangle(int x, int y, int x0, int y0, int x1, int y1, int x2, int y2)
{
    // Calculate vectors
    int v0x = x2 - x0;
    int v0y = y2 - y0;
    int v1x = x1 - x0;
    int v1y = y1 - y0;
    int v2x = x - x0;
    int v2y = y - y0;

    // Calculate dot products
    int dot00 = v0x * v0x + v0y * v0y;
    int dot01 = v0x * v1x + v0y * v1y;
    int dot02 = v0x * v2x + v0y * v2y;
    int dot11 = v1x * v1x + v1y * v1y;
    int dot12 = v1x * v2x + v1y * v2y;

    // Calculate barycentric coordinates
    int denom = dot00 * dot11 - dot01 * dot01;
    if (denom == 0)
    {
        return false; // Triangle is degenerate
    }
    float invDenom = 1.0f / denom;
    float u = (dot11 * dot02 - dot01 * dot12) * invDenom;
    float v = (dot00 * dot12 - dot01 * dot02) * invDenom;

    // Check if point is in triangle
    return (u >= 0) && (v >= 0) && (u + v <= 1);
}

__device__ void rasterizeTriangle(int x0, int y0, int x1, int y1, int x2, int y2, unsigned char *color, uint8_t *bitmap, int width, int height)
{
    int minX = min(x0, min(x1, x2));
    int minY = min(y0, min(y1, y2));
    int maxX = max(x0, max(x1, x2));
    int maxY = max(y0, max(y1, y2));

    // Clipping to the bitmap bounds
    minX = max(minX, 0);
    minY = max(minY, 0);
    maxX = min(maxX, width - 1);
    maxY = min(maxY, height - 1);

    for (int y = minY; y <= maxY; y++)
    {
        for (int x = minX; x <= maxX; x++)
        {
            int idx = (y * width + x) * 4;

            if (isPointInTriangle(x, y, x0, y0, x1, y1, x2, y2))
            {
                bitmap[idx] = color[0];     // B
                bitmap[idx + 1] = color[1]; // G
                bitmap[idx + 2] = color[2]; // R
                bitmap[idx + 3] = color[3]; // A
            }
        }
    }
}

__global__ void drawTrianglesKernel(int *x, int *y, int *indices, int numTriangles, unsigned char *color, uint8_t *bitmap, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numTriangles)
    {
        int idx0 = indices[i * 3];
        int idx1 = indices[i * 3 + 1];
        int idx2 = indices[i * 3 + 2];

        int x0 = x[idx0];
        int y0 = y[idx0];
        int x1 = x[idx1];
        int y1 = y[idx1];
        int x2 = x[idx2];
        int y2 = y[idx2];

        rasterizeTriangle(x0, y0, x1, y1, x2, y2, color, bitmap, width, height);
    }
}

extern "C" __declspec(dllexport) unsigned char *drawPolygonWithTriangles(int *x, int *y, int *indices, int numTriangles, uint8_t *color, uint8_t *bitmap, int width, int height)
{
    uint8_t *d_bitmap;
    uint8_t *d_color;
    int *d_x;
    int *d_y;
    int *d_indices;
    size_t bitmapsize = width * height * 4;
    int vertexcount = numTriangles + 2;

    // Allocate memory on the device
    hipMalloc((void **)&d_bitmap, bitmapsize);
    hipMalloc((void **)&d_color, 4);
    hipMalloc((void **)&d_x, vertexcount * sizeof(int));
    hipMalloc((void **)&d_y, vertexcount * sizeof(int));
    hipMalloc((void **)&d_indices, numTriangles * 3 * sizeof(int));

    // Copy data to device
    hipMemcpy(d_color, color, 4, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, vertexcount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, vertexcount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, indices, numTriangles * 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bitmap, bitmap, bitmapsize, hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 blockDim(256);
    dim3 gridDim((numTriangles + blockDim.x - 1) / blockDim.x);

    // Launch the kernel
    drawTrianglesKernel<<<gridDim, blockDim>>>(d_x, d_y, d_indices, numTriangles, d_color, d_bitmap, width, height);
    hipDeviceSynchronize();

    // Copy the result back to the host
    uint8_t *result_bitmap = (uint8_t *)malloc(bitmapsize);
    hipMemcpy(result_bitmap, d_bitmap, bitmapsize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_bitmap);
    hipFree(d_color);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_indices);

    // Return the resulting bitmap
    return result_bitmap;
}
