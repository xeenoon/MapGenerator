#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include <>
#include <fstream>
#include <stdarg.h>

#define BYTES_PER_PIXEL 4

std::ofstream logFile("C:\\Users\\ccw10\\Downloads\\segfaultlog.txt", std::ios::app);

void logMessage(const char *message, ...)
{
    if (logFile.is_open())
    {
        // Get current time
        std::time_t now = std::time(nullptr);
        std::tm *localTime = std::localtime(&now);

        // Write time to the file
        // logFile << std::asctime(localTime) << ": ";

        // Handle variable arguments
        va_list args;
        va_start(args, message);

        // Print formatted message to a temporary buffer
        char buffer[1024];
        vsnprintf(buffer, sizeof(buffer), message, args);

        // Write formatted message to the file
        logFile << buffer << std::endl;

        va_end(args);
    }
}

// For drawing rocks
__global__ void CudaDrawKernel(
    int rockWidth, int rockHeight, int *topleftXs, int *topleftYs,
    int *bakedrectangleLefts, int *bakedrectangleTops, int *bakedrectangleWidths, int *bakedrectangleHeights,
    int *rockcentreXs, int *rockcentreYs, int shadowdst, int shinedst,
    unsigned char *resultbmp_scan0, unsigned char *rockbmp_scan0,
    unsigned char **bakeddistances_dataScan0s, unsigned char **bakedbounds_dataScan0s,
    int **filters, int resultwidth, int resultheight)
{
    int index = blockIdx.z;
    int x = blockIdx.x * blockDim.x + threadIdx.x + topleftXs[index];
    int y = blockIdx.y * blockDim.y + threadIdx.y + topleftYs[index];

    if (x >= resultwidth || y >= resultheight || x < 0 || y < 0)
        return;

    int resultIndex = x * BYTES_PER_PIXEL + y * resultwidth * BYTES_PER_PIXEL;
    int drawidx = x + y * resultwidth;
    bool inpolygon = false;
    double distance = -1.0;

    int adjustedX = x - bakedrectangleLefts[index];
    int adjustedY = y - bakedrectangleTops[index];
    int checkidx = adjustedX * BYTES_PER_PIXEL + adjustedY * bakedrectangleWidths[index] * BYTES_PER_PIXEL;

    if (adjustedX >= 0 && adjustedY >= 0 && adjustedX < bakedrectangleWidths[index] && adjustedY < bakedrectangleHeights[index])
    {
        if (bakedbounds_dataScan0s[index][checkidx + 2])
        {
            inpolygon = true;
        }
        distance = bakeddistances_dataScan0s[index][checkidx];
    }
    // drawnpixels[drawidx] = 1;

    if (inpolygon)
    {
        int rockIndex = (x % rockWidth) * BYTES_PER_PIXEL + (y % rockHeight) * rockWidth * BYTES_PER_PIXEL;
        for (int i = 0; i < 3; ++i)
        {
            resultbmp_scan0[resultIndex + i] = min(255, filters[index][i] + rockbmp_scan0[rockIndex + i]);
        }
    }
    if (distance <= shadowdst && distance >= 0 && (distance == 0 ? inpolygon : true))
    {
        double shadowFactor = distance / (shadowdst * 3) + (1 - (1.0 / 3.0));
        for (int i = 0; i < 3; ++i)
        {
            resultbmp_scan0[resultIndex + i] = min((uint8_t)(resultbmp_scan0[resultIndex + i] * shadowFactor), 255);
        }
    }
    resultbmp_scan0[resultIndex + 3] = 255;
}

extern "C" __declspec(dllexport) uint8_t *CudaDraw(
    int *rockcentreXs, int *rockcentreYs, int *topleftXs, int *topleftYs, int *bottomrightXs, int *bottomrightYs,
    int *bakedrectangleLefts, int *bakedrectangleTops, int *bakedrectangleWidths, int *bakedrectangleHeights,
    unsigned char **bakeddistances_dataScan0s, unsigned char **bakedbounds_dataScan0s,
    int **filters, unsigned char *resultbmp_scan0, int resultwidth, int resultheight,
    unsigned char *rockbmp_scan0, int rockWidth, int rockHeight, int numItems, int maxrockwidth, int maxrockheight)
{
    logMessage("Called from dll...");
    const int shadowdst = 20;
    const int shinedst = 40;

    size_t imageSize = resultwidth * resultheight * BYTES_PER_PIXEL;
    size_t rockImageSize = rockWidth * rockHeight * BYTES_PER_PIXEL;

    // Allocate memory on the GPU for all variables
    unsigned char *d_resultbmp_scan0;
    int *d_rockcentreXs, *d_rockcentreYs, *d_topleftXs, *d_topleftYs, *d_bottomrightXs, *d_bottomrightYs;
    int *d_bakedrectangleLefts, *d_bakedrectangleTops, *d_bakedrectangleWidths, *d_bakedrectangleHeights;
    unsigned char **d_bakeddistances_dataScan0s, **d_bakedbounds_dataScan0s;
    int **d_filters;
    unsigned char *d_rockbmp_scan0;

    hipMalloc(&d_resultbmp_scan0, imageSize);
    hipMalloc(&d_rockcentreXs, numItems * sizeof(int));
    hipMalloc(&d_rockcentreYs, numItems * sizeof(int));
    hipMalloc(&d_topleftXs, numItems * sizeof(int));
    hipMalloc(&d_topleftYs, numItems * sizeof(int));
    hipMalloc(&d_bottomrightXs, numItems * sizeof(int));
    hipMalloc(&d_bottomrightYs, numItems * sizeof(int));
    hipMalloc(&d_bakedrectangleLefts, numItems * sizeof(int));
    hipMalloc(&d_bakedrectangleTops, numItems * sizeof(int));
    hipMalloc(&d_bakedrectangleWidths, numItems * sizeof(int));
    hipMalloc(&d_bakedrectangleHeights, numItems * sizeof(int));
    hipMalloc(&d_bakeddistances_dataScan0s, numItems * sizeof(unsigned char *));
    hipMalloc(&d_bakedbounds_dataScan0s, numItems * sizeof(unsigned char *));
    hipMalloc(&d_filters, numItems * sizeof(int *));
    hipMalloc(&d_rockbmp_scan0, rockImageSize);
    logMessage("Allocated arrays");

    // Copy the result bitmap to the device
    hipMemcpy(d_resultbmp_scan0, resultbmp_scan0, imageSize, hipMemcpyHostToDevice);

    // Copy data arrays to the device
    hipMemcpy(d_rockcentreXs, rockcentreXs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rockcentreYs, rockcentreYs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_topleftXs, topleftXs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_topleftYs, topleftYs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bottomrightXs, bottomrightXs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bottomrightYs, bottomrightYs, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bakedrectangleLefts, bakedrectangleLefts, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bakedrectangleTops, bakedrectangleTops, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bakedrectangleWidths, bakedrectangleWidths, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bakedrectangleHeights, bakedrectangleHeights, numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_rockbmp_scan0, rockbmp_scan0, rockImageSize, hipMemcpyHostToDevice);
    logMessage("Copied arrays");
    // Copy data pointers to the device and allocate memory for each array
    unsigned char **d_bakeddistances_dataScan0s_copy = new unsigned char *[numItems];
    unsigned char **d_bakedbounds_dataScan0s_copy = new unsigned char *[numItems];
    int **d_filters_copy = new int *[numItems];

    // Allocate memory on the device for each data pointer array
    for (int i = 0; i < numItems; ++i)
    {
        size_t bakedrectanglesize = bakedrectangleWidths[i] * bakedrectangleHeights[i] * BYTES_PER_PIXEL;
        hipMalloc(&d_bakeddistances_dataScan0s_copy[i], bakedrectanglesize);
        hipMemcpy(d_bakeddistances_dataScan0s_copy[i], bakeddistances_dataScan0s[i], bakedrectanglesize, hipMemcpyHostToDevice);

        hipMalloc(&d_bakedbounds_dataScan0s_copy[i], bakedrectanglesize);
        hipMemcpy(d_bakedbounds_dataScan0s_copy[i], bakedbounds_dataScan0s[i], bakedrectanglesize, hipMemcpyHostToDevice);

        hipMalloc(&d_filters_copy[i], sizeof(int));
        hipMemcpy(d_filters_copy[i], filters[i], sizeof(int), hipMemcpyHostToDevice);
    }
    logMessage("Copied 2d arrays");

    hipMemcpy(d_bakeddistances_dataScan0s, d_bakeddistances_dataScan0s_copy, numItems * sizeof(unsigned char *), hipMemcpyHostToDevice);
    hipMemcpy(d_bakedbounds_dataScan0s, d_bakedbounds_dataScan0s_copy, numItems * sizeof(unsigned char *), hipMemcpyHostToDevice);
    hipMemcpy(d_filters, d_filters_copy, numItems * sizeof(int *), hipMemcpyHostToDevice);

    // Allocate memory for the output on the device
    unsigned char *d_output;
    hipMalloc(&d_output, imageSize);
    hipMemcpy(d_output, resultbmp_scan0, imageSize, hipMemcpyHostToDevice); // Copy the old bitmap

    dim3 blockSize(32, 32);
    dim3 gridSize((maxrockwidth + blockSize.x - 1) / blockSize.x, (maxrockheight + blockSize.y - 1) / blockSize.y, numItems);

    CudaDrawKernel<<<gridSize, blockSize>>>(
        rockWidth, rockHeight, d_topleftXs, d_topleftYs,
        d_bakedrectangleLefts, d_bakedrectangleTops, d_bakedrectangleWidths, d_bakedrectangleHeights,
        d_rockcentreXs, d_rockcentreYs, shadowdst, shinedst,
        d_output, d_rockbmp_scan0, d_bakeddistances_dataScan0s, d_bakedbounds_dataScan0s,
        d_filters, resultwidth, resultheight);

    hipDeviceSynchronize();

    // Allocate memory on the host for the result
    uint8_t *h_output = new uint8_t[imageSize];

    // Copy the result back to the host
    hipMemcpy(h_output, d_output, imageSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_output);
    hipFree(d_resultbmp_scan0);
    hipFree(d_rockcentreXs);
    hipFree(d_rockcentreYs);
    hipFree(d_topleftXs);
    hipFree(d_topleftYs);
    hipFree(d_bottomrightXs);
    hipFree(d_bottomrightYs);
    hipFree(d_bakedrectangleLefts);
    hipFree(d_bakedrectangleTops);
    hipFree(d_bakedrectangleWidths);
    hipFree(d_bakedrectangleHeights);
    hipFree(d_rockbmp_scan0);

    for (int i = 0; i < numItems; ++i)
    {
        hipFree(d_bakeddistances_dataScan0s_copy[i]);
        hipFree(d_bakedbounds_dataScan0s_copy[i]);
        hipFree(d_filters_copy[i]);
    }
    delete[] d_bakeddistances_dataScan0s_copy;
    delete[] d_bakedbounds_dataScan0s_copy;
    delete[] d_filters_copy;

    return h_output; // Return the pointer to the host memory
}

// CUDA kernel for normal mapping
__global__ void ApplyNormalMapKernel(
    unsigned char *original, unsigned char *normal,
    unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int idx = (x + y * width) * 4;

        // Pointers to the color values
        unsigned char *originalcolor = original + idx;
        unsigned char *normalcolor = normal + idx;

        // Convert normal color to a vector
        float nx = (normalcolor[2] / 255.0f) * 2.0f - 1.0f;
        float ny = (normalcolor[1] / 255.0f) * 2.0f - 1.0f;
        float nz = (normalcolor[0] / 255.0f) * 2.0f - 1.0f;

        // Simple light direction (from top-left)
        float lx = 0.5f;
        float ly = -0.5f;
        float lz = 1.0f;

        // Normalize the light direction
        float length = sqrtf(lx * lx + ly * ly + lz * lz);
        lx /= length;
        ly /= length;
        lz /= length;

        // Compute the dot product of the normal and light direction
        float dot = nx * lx + ny * ly + nz * lz;
        dot = fmaxf(0.0f, dot); // Clamp to [0, 1]

        // Apply the dot product to the original color to get the shaded color
        output[idx + 2] = (unsigned char)(originalcolor[2] * dot);
        output[idx + 1] = (unsigned char)(originalcolor[1] * dot);
        output[idx + 0] = (unsigned char)(originalcolor[0] * dot);
        output[idx + 3] = originalcolor[3]; // Preserve alpha channel
    }
}

extern "C" __declspec(dllexport) void ApplyNormalMap(
    unsigned char *original, unsigned char *normal,
    unsigned char *output, int width, int height)
{
    unsigned char *d_original, *d_normal, *d_output;
    size_t imageSize = width * height * 4 * sizeof(unsigned char);

    hipMalloc(&d_original, imageSize);
    hipMalloc(&d_normal, imageSize);
    hipMalloc(&d_output, imageSize);

    hipMemcpy(d_original, original, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_normal, normal, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    ApplyNormalMapKernel<<<gridSize, blockSize>>>(d_original, d_normal, d_output, width, height);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_original);
    hipFree(d_normal);
    hipFree(d_output);
}
