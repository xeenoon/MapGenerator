﻿#include <stdio.h>
#include "hip/hip_runtime.h"


#define SIZE 1024

__global__ void VectorAddKernel(int* a, int* b, int* c, int n)
{
    int i = threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

extern "C" __declspec(dllexport) int* ExtVectorAdd(int* a, int* b, int n)
{
    int *d_a, *d_b, *d_c;
    int* h_c = new int[n]; // Allocate host memory for result

    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    VectorAddKernel<<<1, n>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return h_c; // Return host memory pointer
}

extern "C" __declspec(dllexport) void FreeMemory(int* ptr)
{
    delete[] ptr; // Free host memory
}
